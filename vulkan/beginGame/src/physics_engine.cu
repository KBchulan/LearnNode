#include "hip/hip_runtime.h"
#include "physics_engine.hpp"
#include <hip/hip_runtime.h>

__global__ void updatePositions(glm::vec2* positions, glm::vec2* velocities, float deltaTime, int numObjects) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numObjects) {
        positions[idx] += velocities[idx] * deltaTime;
        
        // 简单的重力效果
        velocities[idx].y -= 9.81f * deltaTime;
        
        // 边界碰撞检测
        if (positions[idx].y < -1.0f) {
            positions[idx].y = -1.0f;
            velocities[idx].y = -velocities[idx].y * 0.8f;
        }
    }
}

void PhysicsEngine::init(int numObjects) {
    positions.resize(numObjects);
    velocities.resize(numObjects);
    
    // 初始化位置和速度
    for (int i = 0; i < numObjects; i++) {
        positions[i] = glm::vec2(0.0f);
        velocities[i] = glm::vec2(1.0f, 2.0f);
    }
}

void PhysicsEngine::update(float deltaTime) {
    glm::vec2 *d_positions, *d_velocities;
    
    // 分配GPU内存
    hipMalloc(&d_positions, positions.size() * sizeof(glm::vec2));
    hipMalloc(&d_velocities, velocities.size() * sizeof(glm::vec2));
    
    // 复制数据到GPU
    hipMemcpy(d_positions, positions.data(), positions.size() * sizeof(glm::vec2), hipMemcpyHostToDevice);
    hipMemcpy(d_velocities, velocities.data(), velocities.size() * sizeof(glm::vec2), hipMemcpyHostToDevice);
    
    // 启动CUDA核心
    int blockSize = 256;
    int numBlocks = (positions.size() + blockSize - 1) / blockSize;
    updatePositions<<<numBlocks, blockSize>>>(d_positions, d_velocities, deltaTime, positions.size());
    
    // 复制结果回CPU
    hipMemcpy(positions.data(), d_positions, positions.size() * sizeof(glm::vec2), hipMemcpyDeviceToHost);
    hipMemcpy(velocities.data(), d_velocities, velocities.size() * sizeof(glm::vec2), hipMemcpyDeviceToHost);
    
    // 释放GPU内存
    hipFree(d_positions);
    hipFree(d_velocities);
}

std::vector<glm::vec2> PhysicsEngine::getPositions() const {
    return positions;
} 